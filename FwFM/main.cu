#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

#define N_FEATURE 1024
#define N_FIELD 16
#define DIM_FEAT 4
#define INPUT_CNT 1024

__global__ void FwFM(float *result, float *feature, int *input_data, float *interaction_weight)
{
    int data_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // only consider second order interaction.
    float res = 0;
    for (int i = 0; i < N_FIELD; i++)
    {
        for (int j = i + 1; j < N_FIELD; j++)
        {
            int didx = input_data[data_idx * N_FIELD + i];
            int djdx = input_data[data_idx * N_FIELD + j];

            float sum = 0;
            for (int k = 0; k < DIM_FEAT; k++)
            {
                sum += feature[didx * DIM_FEAT + k] * feature[djdx * DIM_FEAT + k];
            }
            res += interaction_weight[i * N_FIELD + j] * sum;
        }
    }
    result[data_idx] = res;
    // int local_input_data[N_FIELD];
    // float local_feature[N_FIELD][DIM_FEAT];

    // int bx = blockIdx.x;
    // int by = blockIdx.y;
    // int tx = threadIdx.x;
    // int ty = threadIdx.y;
    // int col = bx * blockDim.x + tx;
    // int row = by * blockDim.y + ty;

    // for (int i = 0; i < N_FIELD; i++)
    // {
    //     local_input_data[i] = input_data[col * N_FIELD + i];
    //     for (int j = 0; j < DIM_FEAT; j++)
    //         local_feature[i][j] = feature[local_input_data[i] * DIM_FEAT + j];
    // }

    // // only consider second order interaction.
    // float res = 0;
    // for (int i = 0; i < N_FIELD; i++)
    // {
    //     for (int j = i + 1; j < N_FIELD; j++)
    //     {
    //         float sum = 0;
    //         for (int k = 0; k < DIM_FEAT; k++)
    //         {
    //             sum += local_feature[i][k] * local_feature[j][k];
    //         }
    //         res += interaction_weight[i * N_FIELD + j] * sum;
    //     }
    // }
    // result[col] = res;
}

float rand_float()
{
    return (float)rand() / (float)RAND_MAX;
}

int main()
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    cout << "Shared Memory Per Block : " << prop.sharedMemPerBlock << endl;
    cout << "Threads Per Block : " << prop.maxThreadsPerBlock << endl;
    cout << "Regs Per Block : " << prop.regsPerBlock << endl;
    cout << "Multi Processor Count : " << prop.multiProcessorCount << endl;
    cout << "Max Threads : " << prop.maxThreadsDim[0] << endl;

    // set random seed
    srand(1);

    float *feature;
    float *interaction_weight;

    int *input_data;
    float *result;

    // set feature

    hipMallocManaged(&feature, N_FEATURE * DIM_FEAT * sizeof(float));
    hipMallocManaged(&interaction_weight, N_FIELD * N_FIELD * sizeof(float));

    // randomly set feature
    for (int i = 0; i < N_FEATURE; i++)
    {
        for (int j = 0; j < DIM_FEAT; j++)
        {
            feature[i * DIM_FEAT + j] = rand_float();
        }
    }

    // randomly set interaction weight
    for (int i = 0; i < N_FIELD; i++)
    {
        for (int j = i + 1; j < N_FIELD; j++)
        {
            interaction_weight[i * N_FIELD + j] = rand_float();
        }
    }

    // set input data

    hipMallocManaged(&input_data, INPUT_CNT * N_FIELD * sizeof(int));

    // randomly set input data.
    for (int i = 0; i < INPUT_CNT * N_FIELD; i++)
    {
        input_data[i] = rand() % N_FEATURE;
    }

    hipMallocManaged(&result, INPUT_CNT * sizeof(float));

    // run, measure time

    dim3 blocks(64);
    dim3 threads(32);
    float start = clock();
    for (int i = 0; i < 10000; i++)
    {
        FwFM<<<blocks, threads>>>(result, feature, input_data, interaction_weight);
        hipDeviceSynchronize();
    }
    float end = clock();
    float time = (end - start) / CLOCKS_PER_SEC;

    printf("time: %f\n", time);

    // check result
    printf("Output:\n");
    for (int i = 0; i < 10; i++)
    {
        printf("result %d, %f\n", i, result[i]);
    }
    printf("result %d, %f\n", INPUT_CNT - 1, result[INPUT_CNT - 1]);
    printf("result %d, %f\n", INPUT_CNT, result[INPUT_CNT]);
    printf("result %d, %f\n", 2 * INPUT_CNT - 1, result[2 * INPUT_CNT - 1]);

    hipFree(feature);
    hipFree(interaction_weight);
    hipFree(input_data);
    hipFree(result);
    return 0;
}